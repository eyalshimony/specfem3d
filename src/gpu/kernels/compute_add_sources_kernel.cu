#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void compute_add_sources_kernel(realw* accel,
                                           int* d_ibool,
                                           realw* sourcearrays,
                                           field* stf_pre_compute,
                                           int myrank,
                                           int* islice_selected_source,
                                           int* ispec_selected_source,
                                           int* ispec_is_elastic,
                                           int NSOURCES) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;

  int isource  = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int ispec,iglob;

  if (isource < NSOURCES) { // when NSOURCES > 65535, but mod(nspec_top,2) > 0, we end up with an extra block.

    if (myrank == islice_selected_source[isource]) {

      ispec = ispec_selected_source[isource]-1;

      if (ispec_is_elastic[ispec]) {

        iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

        realw stf = (realw) stf_pre_compute[isource];

        realw stf_x = sourcearrays[INDEX5(NSOURCES,NDIM,NGLLX,NGLLX,isource,0,i,j,k)] * stf;
        realw stf_y = sourcearrays[INDEX5(NSOURCES,NDIM,NGLLX,NGLLX,isource,1,i,j,k)] * stf;
        realw stf_z = sourcearrays[INDEX5(NSOURCES,NDIM,NGLLX,NGLLX,isource,2,i,j,k)] * stf;

        atomicAdd(&accel[iglob*3],stf_x);
        atomicAdd(&accel[iglob*3+1],stf_y);
        atomicAdd(&accel[iglob*3+2],stf_z);
      }
    }
  }

}


