#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void add_sources_el_SIM_TYPE_2_OR_3_kernel(realw* accel,
                                                      int nrec,
                                                      int it,
                                                      int NSTEP_BETWEEN_ADJSRC,
                                                      field* source_adjoint,
                                                      realw* xir_store,
                                                      realw* etar_store,
                                                      realw* gammar_store,
                                                      int* d_ibool,
                                                      int* ispec_is_elastic,
                                                      int* ispec_selected_recloc,
                                                      int nadj_rec_local) {

  int irec_local = blockIdx.x + gridDim.x*blockIdx.y;

  if (irec_local < nadj_rec_local) { // when nrec > 65535, but mod(nspec_top,2) > 0, we end up with an extra block.

    int ispec = ispec_selected_recloc[irec_local]-1;

    if (ispec_is_elastic[ispec]){
      int i = threadIdx.x;
      int j = threadIdx.y;
      int k = threadIdx.z;

      int iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      realw hxir    = xir_store[INDEX2(NGLLX,i,irec_local)];
      realw hetar   = etar_store[INDEX2(NGLLX,j,irec_local)];
      realw hgammar = gammar_store[INDEX2(NGLLX,k,irec_local)];

      realw lagrange =   hxir * hetar * hgammar ;

      realw source_adj_x = (realw) source_adjoint[INDEX3(NDIM,nadj_rec_local,0,irec_local,it)];
      realw source_adj_y = (realw) source_adjoint[INDEX3(NDIM,nadj_rec_local,1,irec_local,it)];
      realw source_adj_z = (realw) source_adjoint[INDEX3(NDIM,nadj_rec_local,2,irec_local,it)];

      realw stf_x = source_adj_x * lagrange;
      realw stf_y = source_adj_y * lagrange;
      realw stf_z = source_adj_z * lagrange;

      // atomic operations are absolutely necessary for correctness!
      atomicAdd(&accel[iglob*3],stf_x);
      atomicAdd(&accel[iglob*3+1],stf_y);
      atomicAdd(&accel[iglob*3+2],stf_z);
    } // ispec_is_elastic
  }
}


