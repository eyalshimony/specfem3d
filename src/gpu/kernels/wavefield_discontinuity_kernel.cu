#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void add_acceleration_discontinuity_kernel(
                                  realw_const_p accel_wd,
                                  realw_const_p mass_in_wd,
                                  const int* boundary_to_iglob_wd,
                                  const int size, realw* accel
                                  ) {
  int id = threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x;
  int iglob = boundary_to_iglob_wd[id] - 1;
  realw mass_in = mass_in_wd[id];
  if (id < size) {
    accel[iglob*3] = accel[iglob*3] - accel_wd[id*3] * mass_in;
    accel[iglob*3 + 1] = accel[iglob*3 + 1] - accel_wd[id*3 + 1] * mass_in;
    accel[iglob*3 + 2] = accel[iglob*3 + 2] - accel_wd[id*3 + 2] * mass_in;
  }
}

__global__ void add_traction_discontinuity_kernel(
                                  realw_const_p traction_wd,
                                  const int* face_ispec_wd,
                                  const int* face_ijk_wd,
                                  realw_const_p face_jacobian2Dw_wd,
                                  const int* d_ibool,
                                  const int size, realw* accel) {
  int igll = threadIdx.x;
  int iface_wd = blockIdx.x + gridDim.x*blockIdx.y;
  int i, j, k, ispec, iglob;
  realw jacobianw;
  if (iface_wd < size) {
    ispec = face_ispec_wd[iface_wd] - 1;
    i = face_ijk_wd[INDEX3(NDIM,NGLL2,0,igll,iface_wd)]-1;
    j = face_ijk_wd[INDEX3(NDIM,NGLL2,1,igll,iface_wd)]-1;
    k = face_ijk_wd[INDEX3(NDIM,NGLL2,2,igll,iface_wd)]-1;

    iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

    jacobianw = face_jacobian2Dw_wd[INDEX2(NGLL2,igll,iface_wd)];
    atomicAdd(&accel[iglob*3],  traction_wd[INDEX3(NDIM,NGLL2,0,igll,iface_wd)] * jacobianw);
    atomicAdd(&accel[iglob*3+1],  traction_wd[INDEX3(NDIM,NGLL2,1,igll,iface_wd)] * jacobianw);
    atomicAdd(&accel[iglob*3+2],  traction_wd[INDEX3(NDIM,NGLL2,2,igll,iface_wd)] * jacobianw);
  }
}
